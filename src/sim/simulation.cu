#include "hip/hip_runtime.h"
//Team Cosmosis

#include "simulation.h"
#include "simbody.cu"
#include "..\timer.h"
#include <hip/hip_runtime.h>

#include <cstdint>

BodyArray MakeArray(thrust::device_vector<SimBody>& arr)
{
    BodyArray ba = { thrust::raw_pointer_cast(&arr[0]), arr.size() };
    return ba;
}

void __global__ SimCalc(BodyArray a)
{
    int_fast32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < a.size) {
		const _T G = 6.67384f * pow(10.0f, -11.0f);
		//precompute positions at index
		const _T px = a.array[idx].Position.x;
		const _T py = a.array[idx].Position.y;
		//mass at the index
		const _T M_idx = G*a.array[idx].Mass;

		a.array[idx].Force = vec2_t();
        for (int_fast32_t j(0); j != a.size; ++j) {
            if (idx != j) {
				_T dx = a.array[j].Position.x - px;
				_T dy = a.array[j].Position.y - py;
				_T r = sqrt(dx*dx + dy*dy);
				_T F = (M_idx*a.array[j].Mass)/(r*r);
				a.array[idx].Force.x += F * (dx / r);
				a.array[idx].Force.y += F * (dy / r);
            }
        }
    }
}

void __global__ SimTick(BodyArray a, _T dt)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < a.size)
    {
		_T mass = a.array[idx].Mass;
		a.array[idx].Velocity.x += dt * (a.array[idx].Force.x / mass);
		a.array[idx].Velocity.y += dt * (a.array[idx].Force.y / mass);
		a.array[idx].Position.x += dt * a.array[idx].Velocity.x;
		a.array[idx].Position.y += dt * a.array[idx].Velocity.y;
    }
}

Simulation::Simulation(void) : sampleCount_(-1), numBlocks_(0), numThreads_(0) { }

Simulation &Simulation::GetInstance(void)
{
    static Simulation self;
    return self;
}

int Simulation::Setup(int argc, char *argv[])
{
    if (argc < 2)
    {
        std::cout << "Invalid number of arguments." << std::endl <<
                  "Usage: " << argv[0] << " [num bodies] <max samples>" << std::endl;
        return 1;
    }
    if (argc == 3)
    {
        int do_samples = atoi(argv[2]);
        if (do_samples < 1 || do_samples > 10240)
        {
            std::cout << "** Invalid number of samples, must be between 1 and 10240. **" << std::endl;
            return 1;
        }
        sampleCount_ = do_samples;
    }
    int num_bodies = atoi(argv[1]);
    if (num_bodies < 0 || num_bodies > 16384)
    {
        std::cout << "** Invalid number of bodies, must be between 1 and 16384. **" << std::endl;
        return 1;
    }
    std::cout << "Setting up " << num_bodies << " bodies." << std::endl;
    srand((unsigned)time(NULL));

	bodies_.reserve(num_bodies);
    for (unsigned i = 0; i < (unsigned)num_bodies; ++i)
        bodies_.push_back(SimBody(
                              random(1.0E11f, 3.0E11f),
                              random(-6.0E11f, 9.0E11f),
                              random(-1000.0f, 1000.0f),
                              random(-1000.0f, 1000.0f),
                              random(1.0E9f, 1.0E31f)));
    std::cout << "Configuring CUDA... " << std::endl;

    int device;
    hipDeviceProp_t prop;
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        std::cout << hipGetErrorString(err) << std::endl;
        return 1;
    }
    err = hipGetDevice(&device);
    if (err != hipSuccess)
    {
        std::cout << "Error getting CUDA device... aborting." << std::endl;
        return 1;
    }
    err = hipGetDeviceProperties(&prop, device);
    if (err == hipErrorInvalidDevice)
    {
        std::cout << "Invalid CUDA device found... aborting." << std::endl;
        return 1;
    }
	int maxBlocks = prop.major > 2 ? 16 : 8;

	numThreads_ = prop.maxThreadsPerMultiProcessor / maxBlocks;
	numBlocks_ = (bodies_.size() + numThreads_ - 1) / numThreads_;

	numThreads_ = (numThreads_ + 1) & ~1;

    std::cout << "CUDA setup complete. Using:" << std::endl <<
              "\tBlocks: " << numBlocks_ << std::endl <<
              "\tThreads: " << numThreads_ << std::endl <<
			  "\tMax Blocks: " << ((prop.major > 2) ? 16 : 8) << std::endl <<
			  "\tMax Resident Threads: " << prop.maxThreadsPerMultiProcessor << std::endl;
    std::cout << "Completed setup... computing... " << std::endl;
    return 0;
}

int Simulation::Run(void)
{
    running_ = true;
    float timeStep = 25000.0f;

    unsigned sample = 0;

    Timer timer;
    timer.start();
    if (sampleCount_ > 0)
        std::cout << "Running test for " << sampleCount_ << " samples..." << std::endl;

		
    BodyArray arr = MakeArray(bodies_);

    while (running_)
    {
		SimCalc <<<numBlocks_, numThreads_>>>(arr);
		//SimCalc <<<numBlocks_, numThreads_>>>(arr);
		hipDeviceSynchronize();
		SimTick <<<numBlocks_, numThreads_>>>(arr, timeStep);
		//SimTick <<<numBlocks_, numThreads_>>>(arr, timeStep);
        hipDeviceSynchronize();

        ++sample;

        if (sampleCount_ < 0)
        {
            if (sample % SAMPLE_RATE == 0)
            {
                std::cout << sample << " Samples taken avg. " << std::fixed
                          << float(timer.getElapsedTimeInMilliSec() / (float)sample)
                          << " ms/sample. " << float((float)sample / timer.getElapsedTimeInSec())
                          << " samples/sec. " << std::endl;
            }
            if (timer.getElapsedTimeInSec() > TIME_TO_LIVE)
                running_ = false;
        }
        else if (sample == sampleCount_)
        {
            running_ = false;
        }
    }
	hipDeviceReset();	

    timer.stop();
    std::cout << sample << " Samples taken avg. " << std::fixed
              << float(timer.getElapsedTimeInMilliSec() / (float)sample)
              << " ms/sample. " << float((float)sample / timer.getElapsedTimeInSec())
              << " samples/sec. " << std::endl;
    if (sampleCount_ > 0)
        std::cout << "Total elapsed time: " << timer.getElapsedTimeInSec() << " seconds." << std::endl;
    std::cout << "Completed the test with " << sample << " samples. Press any key to exit." << std::endl;	
    return 0;
}