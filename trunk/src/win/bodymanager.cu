#include "hip/hip_runtime.h"
#pragma once
#include <SFML/Graphics.hpp>
#include <SFML/Graphics/RenderWindow.hpp>
#include <vector>
#include "body.cu"
#include <fstream>
#include <cmath>
#include <vector>
#include <thrust/device_vector.h>
#include <hip/hip_runtime.h>
#include "imagemanager.h"


typedef struct
{
    Body *array;
    unsigned size;
} BodyArray;

BodyArray MakeArray(thrust::device_vector<Body> &arr)
{
    BodyArray ba = 
    { thrust::raw_pointer_cast(&arr[0]), arr.size() };
    return ba;
}

void __global__ RenderK(BodyArray bodies) {
/*
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	Body& body = bodies[idx];
	body.SetSpritePosition(body.Position.x*zoomLevel_/solarRadius_, body.Position.y*zoomLevel_/solarRadius_);
*/
}

void __global__ TickTop(BodyArray bodies) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < bodies.size) {
		bodies.array[idx].ResetForce();

		for(size_t j = 0; j < bodies.size; ++j)
			if(idx != j)
				bodies.array[idx].AddForce(bodies.array[j]);
	}
}

void __global__ TickBottom(BodyArray bodies, float time) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < bodies.size) {
		bodies.array[idx].Tick(time);
}



class BodyManager : sf::NonCopyable {
	BodyManager(void) 
		: imageManager_(ImageManager::GetInstance()) {
		zoomLevel_ = 128;
		srand((unsigned int)time(NULL)); //take out int      *srand((unsigned)time(NULL));
	}

	~BodyManager(void) {
		bodies_.clear();
	}

	thrust::device_vector<Body> bodies_;
    BodyArray arr_;
	ImageManager& imageManager_;
	sf::RenderWindow* app_;
	double solarRadius_;	
	size_t zoomLevel_;

    unsigned numBlocks_;
    unsigned numThreads_;
public:
	static BodyManager& GetInstance() {
		static BodyManager self;
		return self;
	}

	void Render( void ) {
	/*
		RenderK<<<#,#>>>(bodies_);
		hipDeviceSynchronize();

		size_t size = bodies_.size();
		for(size_t i = 0; i < size; ++i) {
			//Could put this into a for loop to draw (on the host)
			app_->Draw(body);		
		}
	*/
	}

    void Tick(float timeStep) {
        TickTop<<<numBlocks_, numThreads_>>>(arr_);
        if (hipDeviceSynchronize() != hipSuccess)
            std::cout << "Error Tick!" << std::endl;
        TickBottom<<<numBlocks_, numThreads_>>>(arr_, timeStep);
        if (hipDeviceSynchronize() != hipSuccess)
            std::cout << "Error Tick!" << std::endl;
    }

	bool Init(int count, double radius, sf::RenderWindow* app) {	
		if(app == NULL || count <= 0 || radius <= 0) return false;

		bodies_.clear();
		bodies_.reserve(count*16);
		solarRadius_ = radius;
		app_ = app;

		return true;
	}

	bool InitFromFile(const char* fileName, sf::RenderWindow* app) {
		char fileStr[260];
		size_t count = 0;
		double radius = 0.0, rx, ry, vx, vy, m;
	
		//Bad parameters
		if(fileName == NULL || app == NULL) return false;

		//Set the render window
		app_ = app;
		//Remove previous loads
		bodies_.clear();

		FILE* file = fopen(fileName, "r");
	
		//File doesn't exist
		if(file == NULL) return false;

		//Read the count of planets and the radius of the universe.
		fscanf(file, "%d\n", &count);
		fscanf(file, "%lf\n", &radius);

		//No count or radius is bad.
		if(count <= 0 || radius <= 0.0) {		
			fclose(file);
			return false;
		}

		//Set the solar radius
		solarRadius_ = radius;
		//Reserve count amount of items, for faster adding.
		bodies_.reserve(count);

        // ------ kernel launch configurations starts here
        int dev;
        hipError_t error;
        hipDeviceProp_t prop;

        if (hipGetDevice(&dev) != hipSuccess){
            std::cout << "Error 1" << std::endl;
            return 1;
        }

        if (hipGetDeviceProperties(&prop, dev) != hipSuccess){
            std::cout << "Error 1" << std::endl;
            return 1;
        }

        numThreads_ = prop.maxThreadsDim[0];
        numBlocks_ = (count + numThreads_ - 1) / numThreads_;

		//Add the bodies
		//Make this part into a kernel ?
		for(size_t i = 0; i < count; ++i) {		
			fscanf(file, "%lf %lf %lf %lf %lf %s\n", &rx, &ry, &vx, &vy, &m, &fileStr);		
			AddBody(Body(imageManager_.GetImage(fileStr), rx, ry, vx, vy, m));
		}

        arr_ = MakeArray(bodies_);

		return !fclose(file);
	}

	CUDA_CALLABLE_MEMBER void AddBody(const Body& body) {
		bodies_.push_back(body);
	} 

	void ClearBodies() {
		bodies_.clear();
	}

	double GetSolarRadius() const {
		return solarRadius_;
	}

	void IncreaseZoom() {
		zoomLevel_ <<= 1;
	}

	void DecreaseZoom() {
		zoomLevel_ >>= 1;
		if(zoomLevel_ == 0)
			zoomLevel_ = 1;
	}

	int ZoomLevel() const {
		return zoomLevel_;
	}

    unsigned getBlocks() const {
        return numBlocks_;
    }

    unsigned getThreads() const {
        return numThreads_;
    }
};