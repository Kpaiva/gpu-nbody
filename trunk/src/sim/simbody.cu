#include "hip/hip_runtime.h"
//Team Cosmosis

#pragma once
#ifndef BODY_H
#define BODY_H

#include "../common.h"
#include <complex>
#include <hip/hip_runtime.h>

template <typename T>
class _SimBody {
private:
#if IS_TESTING
	T ComputeGC(T m1, T m2, T d) {
		const T G = 6.67384f * pow(10.0f, -11.0f);
		return (G*m1*m2)/(d*d);
	}
#endif
public:
	T Mass;
	vec2_t Force;
	vec2_t Velocity;
	vec2_t Position;

	_SimBody(T px = 0.0f, T py = 0.0f, T vx = 0.0f, T vy = 0.0f, T mass = 0.0f) :
		Mass(mass), Velocity(vx,vy), Position(px,py), Force() { }

#if IS_TESTING
	void Tick(T dt) {
		Velocity.x += dt * (Force.x / Mass);
		Velocity.y += dt * (Force.y / Mass);
		Position.x += dt * Velocity.x;
		Position.y += dt * Velocity.y;
	}

	void AddForce(const SimBody& b) {
		T dx = b.Position.x - Position.x;
		T dy = b.Position.y - Position.y;
		T r = sqrt(dx*dx + dy*dy);
		T F = ComputeGC(Mass, b.Mass, r);
		Force.x += F * (dx / r);
		Force.y += F * (dy / r);
	}

	void ResetForce() {
		Force = vec2_t();
	}
#endif
};

#endif //BODY_H
